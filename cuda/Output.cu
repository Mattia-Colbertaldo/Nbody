#include "hip/hip_runtime.h"
#include "Output.cuh"
#include <iostream>
#include <thrust/device_vector.h>


// =================
// Helper Functions
// =================

/*
    classe che gestisce l’output su file. Ha attributi che identificano il nome del file di output 
    e ogni quanti step fare output su file e const reference ai vettori con le particelle.
    La classe gestisce dietro le quinte l’output, in particolare avrà un metodo save che controlla implicitamente a quale step siamo,
    se a questo step va effettuato l’output e se la risposta è affermativa solo il rank 0 scrive su file. 
*/

void Output::save(std::ofstream& fsave, const std::unique_ptr<AllParticles> & parts, const double size, const int& nsteps){
    
  static bool first = true;

  if (first) {
      fsave << parts->num_parts << " " << size << " " << nsteps << "\n";
      first = false;
  }

  // Opzione 1:
  thrust::copy(parts->x.begin(), parts->x.end(), parts->x_h.begin());
  thrust::copy(parts->y.begin(), parts->y.end(), parts->y_h.begin());
  thrust::copy(parts->z.begin(), parts->z.end(), parts->z_h.begin());
  // Opzione 2:
  // x_h = x;
  // y_h = y;
  // z_h = z;




  hipDeviceSynchronize();
  for(size_t i = 0; i < parts->num_parts; i++){
    // TODO X_H
        fsave <<  parts->x_h[i] << " " << parts->y_h[i] << " " << parts->z_h[i] << std::endl;
  }
  hipDeviceSynchronize();
};

void Output::save_output(std::ofstream& fsave, const int savefreq, const std::unique_ptr<AllParticles> & parts , const int& step,  const int& nsteps, const double & size){
    // TODO FIX
    // thrust::copy(x.begin(), x.end(), x_h.begin());
    // thrust::copy(y.begin(), y.end(), y_h.begin());
    // thrust::copy(z.begin(), z.end(), z_h.begin());

    // save(fsave, parts, size, nsteps);
    
    thrust::copy(parts->x.begin(), parts->x.end(), &bufferx[num_parts*step]);
    thrust::copy(parts->y.begin(), parts->y.end(), &buffery[num_parts*step]);
    thrust::copy(parts->z.begin(), parts->z.end(), &bufferz[num_parts*step]);

    hipDeviceSynchronize();
    if(step > 0){
        if (step%10 == 0){
        fflush(stdout);
        printf("[ %d ]\r", (int)(step*100/nsteps));
        }
    }
    if(step == nsteps - 1){
        std::cout << "Retrieving data from GPU: ";
        long t = clock();
        thrust::copy(bufferx.begin(), bufferx.end(), host_bufferx.begin());
        thrust::copy(buffery.begin(), buffery.end(), host_buffery.begin());
        thrust::copy(bufferz.begin(), bufferz.end(), host_bufferz.begin());
        std::cout << ((clock() - t)*MS_PER_SEC)/CLOCKS_PER_SEC << " ms" << std::endl;
        std::cout << "Saving: ";
        t = clock();
        for(size_t i = 0; i < parts->num_parts*nsteps; i++){
            if (i%10 == 0){
            fflush(stdout);
            printf("\r");
            printf("Saving: [ %d ]\r", (int)(i*100/(parts->num_parts*nsteps)));
            }
            fsave <<  host_bufferx[i] << " " << host_buffery[i] << " " << host_bufferz[i] << std::endl;
        }
        std::cout << "Saving: " << ((clock() - t)*MS_PER_SEC)/CLOCKS_PER_SEC << " ms" << std::endl;

    }
};
