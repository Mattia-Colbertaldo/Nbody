#include "hip/hip_runtime.h"
#include "common.cuh"
#include "AllParticles.cuh"

#include <memory>
#include <stdexcept>
#include <cmath>

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <sm_60_atomic_functions.h>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/execution_policy.h>






__global__ void move_kernel(double* dx, double* dy, double* dz,
                        double* dvx, double* dvy, double* dvz,
                        double* dax, double* day, double* daz, const double size, const int num_parts){
    // double size = dsize[0];
    // int num_parts = dnum_parts[0];
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i>=num_parts) return;
    dvx[i] += dax[i] * dt;
    dvy[i] += day[i] * dt;
    dvz[i] += daz[i] * dt;
    dx[i] += dvx[i] * dt;
    dy[i] += dvy[i] * dt;
    dz[i] += dvz[i] * dt;

    // Bounce from walls
    while (dx[i] < 0 || dx[i] > size) {
        dx[i] = (dx[i] < 0 ? -dx[i] : 2 * size - dx[i]);
        dvx[i] = -dvx[i];
    }

    while (dy[i] < 0 || dy[i] > size) {
        dy[i] = (dy[i] < 0 ? -dy[i] : 2 * size - dy[i]);
        dvy[i] = -dvy[i];
    }

    while (dz[i] < 0 || dz[i] > size) {
        dz[i] = (dz[i] < 0 ? -dz[i] : 2 * size - dz[i]);
        dvz[i] = -dvz[i];
    }
};



__global__ void ResetAcc(double* ax, double* ay, double* az, const int num_parts){
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i>=num_parts) return;
  ax[i] = 0.0;
  ay[i] = 0.0;
  az[i] = 0.0;
};

void AllParticles::ResetAccelerations(){
  ResetAcc<<<ceil(num_parts/th_per_block), th_per_block>>>(dax, day, daz, num_parts);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
      return;
    }
}

void AllParticles::init(){
    {
            thrust::default_random_engine rng;
            thrust::uniform_real_distribution<double> dist(0.0, size);
            thrust::uniform_real_distribution<double> dist1(-1.0, 1.0);

            for(int i=0; i<num_parts; i++){
              x_h[i] = dist(rng);
              y_h[i] = dist(rng);
              z_h[i] = dist(rng);
              vx[i] = dist1(rng);
              vy[i] = dist1(rng);
              vz[i] = dist1(rng);
              //pos[i] = make_double3(dist(rng), dist(rng), dist(rng));
              //vel[i] = make_double3(dist1(rng), dist1(rng), dist1(rng));
              //masses[i] = (dist1(rng) + 1.0);
              masses[i] = dist1(rng)+2.0;
              charges[i] = dist1(rng)*1e-19;
            }

            thrust::copy(x_h.begin(), x_h.end(), x.begin());
            thrust::copy(y_h.begin(), y_h.end(), y.begin());
            thrust::copy(z_h.begin(), z_h.end(), z.begin());

            // TODO mettere inizializzazione di xh e poi copy al vettore trust
            

            hipDeviceSynchronize();
            ResetAccelerations();
            



        }
}


void AllParticles::save(std::ofstream& fsave){
    
  static bool first = true;

  if (first) {
      fsave << num_parts << " " << size << " " << nsteps << "\n";
      first = false;
  }
  //dovrei scrivere x_h[i] per risparmiare tempo ma non funziona. Ci penserò più tardi

  // Opzione 1:
  // thrust::copy(x.begin(), x.end(), x_h.begin());
  // thrust::copy(y.begin(), y.end(), y_h.begin());
  // thrust::copy(z.begin(), z.end(), z_h.begin());
  // Opzione 2:
  // x_h = x;
  // y_h = y;
  // z_h = z;
  hipDeviceSynchronize();
  for(size_t i = 0; i < num_parts; i++){
    // TODO X_H
        fsave <<  x_h[i] << " " << y_h[i] << " " << z_h[i] << std::endl;
  }
  hipDeviceSynchronize();
};

void AllParticles::save_output(std::ofstream& fsave, int step){
    // TODO FIX
    thrust::copy(x.begin(), x.end(), x_h.begin());
    thrust::copy(y.begin(), y.end(), y_h.begin());
    thrust::copy(z.begin(), z.end(), z_h.begin());
    save(fsave);
    hipDeviceSynchronize();
    if(step > 0){
        if (step%10 == 0){
        fflush(stdout);
        printf("[ %d% ]\r", (int)(step*100/nsteps));
        }
    }
};

    // Integrate the ODE

    void AllParticles:: move() {
      move_kernel<<<grid_sizes, block_sizes>>> ( dx, dy, dz,
                                                 dvx, dvy, dvz,
                                                 dax, day, daz, size, num_parts);
                                                 
    };