#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>

#include <thrust/device_vector.h>

/*
 * --- Using Thrust vectors in kernel functions ---
 * This example demonstrates how we can use device vectors (vectors that live
 * in GPU) as arguments to a kernel launched by the CPU.
 *
 * Every time you push a value in the thrust::device_vector it is expensive as
 * it needs to send it individually to GPU. So in this case it is a not very
 * good way to initialize a particle system, but it is good for an example. It
 * would be better to initialize all its (vector) members all at once.
 *
 * The device_vectors are members of the particle system. Remember that Thrust
 * is simply an abstraction that works on CPU. So we cannot send them to a
 * kernel. However what we can do, is to get the GPU pointer to the first
 * element of the vector and send it to the kernel. This is absolutely great.
 *
 * Compilation instructions (in NCCA labs):
 *     nvcc -g -I/usr/include/cuda -x cu main.cpp 
 * 
 * NOTE: Thrust lives under /usr/include/cuda/thrust so with one -I we can find
 * both by prefixing <thrust/...> or just <...> (for cuda headers).
 */

// This is the kernel that is launched from CPU and GPU runs it for each cell
__global__ void kernel(float *positions, float *velocities)
{
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    positions[index] += velocities[index];
    // positions[index] = index; // use this one for debugging the index
}

// The particle system stores the point positions and velocities in
// thrust::vector (not to be confused with a cartesian space vector). Cool thing
// of vectors is that they store the elements contiguously in memory. If we had
// 3 particles in the system the positions thrust::device_vector would look like
//     positions: [particle0posx, particle0posy, particle0posz, particle1posx...]
// Same applies for the velocities. That is a sensible way of storing data if we
// want later OpenGL to render them. I will send an example on how to share a
// Vertex attribute buffer with CUDA later.
class ParticleSystem
{
  public:
    ParticleSystem() : num(0) {};
    void birth_particle();       // pushes back one more particle data to the device_vectors
    void advance_particles();    // launches the kernel that adds velocity to positions
    thrust::device_vector<float> positions;   
    thrust::device_vector<float> velocities;
    unsigned int num;            // number of particles
};

int main(void)
{
  int numParticles = 1024;
  int numSteps = 1;    // understand it as "frames", how many steps in time

  ParticleSystem ps;
  
  // Send new positions and velocities to GPU
  for (int i = 0; i < numParticles; i++)
    ps.birth_particle();
  
  // Particle data lives in GPU now so we call the kernel on them few times!
  // This is great! As we don't have to be retrieving and re-sending, Thrust
  // functionality shines in this step. Great framework.
  for (int i = 0; i < numSteps; i++)
    ps.advance_particles();

  std::cout << std::fixed;
	std::cout << std::setprecision(2);

  // This is gonna be an expensive way of accessing the positions, as for each
  // call to the ::operator[]() we are fetching the item from GPU to CPU. The 
  // right way would be to copy the device_vector into a host_vector like this:
  //    thrust::host_vector<float> host_vector = ps.positions
  // That would do it paralelly, it would transfer all the items from the device
  // vector into the host_vector in a parallel way, but to keep it simply in the
  // code I will not be using it.
  for (int i = 0; i < ps.num; i++)
    std::cout << ps.positions[3*i] << " " << ps.positions[3*i+1] << " " << ps.positions[3*i+2] << std::endl;

  return 0;
}

void ParticleSystem::birth_particle()
{
  positions.push_back(2.0f);
  positions.push_back(2.5f);
  positions.push_back(2.0f);

  velocities.push_back(2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f);
  velocities.push_back(2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f);
  velocities.push_back(2.0f * ((float)rand() / (float)RAND_MAX) - 1.0f);

  num += 1;
}

void ParticleSystem::advance_particles()
{
  // As we cannot send device vectors to the kernel (as device_vector is at
  // the end of the day a GPU structure abstraction in CPU) we have to get the
  // pointer in GPU memory in order for the kernel to know where to start 
  // reading the float arrays from.
  float* d_positions =  thrust::raw_pointer_cast(&positions[0]);
  float* d_velocities = thrust::raw_pointer_cast(&velocities[0]);

  /* This is the way I structured my blocks and threads. I fixed the amount of
   * threads per block to 1024. So to get the amount of blocks we just get the
   * total number of elements in positions and divide it by 1024. We add one in
   * case the division leaves remainder.
   *
   * ┌──────────────────────grid─┬of─blocks─────────────────┬──────────
   * │     block_of_threads      │     block_of_threads     │  
   * │ ┌───┬───┬───────┬──────┐  │ ┌───┬───┬───────┬──────┐ │
   * │ │ 0 │ 1 │ [...] │ 1023 │  │ │ 0 │ 1 │ [...] │ 1023 │ │   ...
   * │ └───┴───┴───────┴──────┘  │ └───┴───┴───────┴──────┘ │
   * └───────────────────────────┴──────────────────────────┴──────────
   */
  unsigned int num_of_elements_per_array = 3 * num;
  unsigned int block_size = 1024;
  unsigned int grid_size = num_of_elements_per_array / block_size + 1;

  std::cout << "Num of elements per array: " << num_of_elements_per_array << std::endl;
  std::cout << "Num of blocks in grid: " << grid_size << std::endl;
  std::cout << "Num of threads per block: " << block_size << std::endl;

  // Launch the kernel! As you can see we are not copying memory from CPU to GPU
  // as you would normally do with hipMemcpy(), as we don't need to! The
  // vectors live in GPU already so we just need to know where they start (GPU
  // pointer) and pass it to the kernel. No need to copy back, we can read from
  // the device vector with the ::operator[]() i.e. positions[2] and that would
  // do all the memory copying for us!
  kernel<<<grid_size,block_size>>>(d_positions, d_velocities);
}